#include "hip/hip_runtime.h"
#include "TextureSimilarityMeasures.hh"

__device__ uint8_t getPixel(Color *image, int x, int y, size_t pitch, int width, int height) {
    if (x < 0 || x >= width || y < 0 || y >= height) {
        return 0;
    }
    Color pixel = image[y * pitch / sizeof(Color) + x];
    return pixel.r / 2 + pixel.g / 2;
}

__global__ void TextureSimilarityKernel(Color *d_img1, size_t pitch1,  Color *d_img2, size_t pitch2, double *d_similarities, size_t pitch, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        uint8_t centerPixel1 = (d_img1[y * pitch1 / sizeof(Color) + x].r + d_img1[y * pitch1 / sizeof(Color) + x].g) / 2;
        uint8_t centerPixel2 = (d_img2[y * pitch2 / sizeof(Color) + x].r + d_img2[y * pitch2 / sizeof(Color) + x].g) / 2;

        uint8_t current_lbpCode1 = 0;
        current_lbpCode1 |= (getPixel(d_img1, x - 1, y - 1, pitch1, width, height) < centerPixel1) << 7;
        current_lbpCode1 |= (getPixel(d_img1, x, y - 1, pitch1, width, height) < centerPixel1) << 6;
        current_lbpCode1 |= (getPixel(d_img1, x + 1, y - 1, pitch1, width, height) < centerPixel1) << 5;
        current_lbpCode1 |= (getPixel(d_img1, x + 1, y, pitch1, width, height) < centerPixel1) << 4;
        current_lbpCode1 |= (getPixel(d_img1, x + 1, y + 1, pitch1, width, height) < centerPixel1) << 3;
        current_lbpCode1 |= (getPixel(d_img1, x, y + 1, pitch1, width, height) < centerPixel1) << 2;
        current_lbpCode1 |= (getPixel(d_img1, x - 1, y + 1, pitch1, width, height) < centerPixel1) << 1;
        current_lbpCode1 |= (getPixel(d_img1, x - 1, y, pitch1, width, height) < centerPixel1);

        uint8_t current_lbpCode2 = 0;
        current_lbpCode2 |= (getPixel(d_img2, x - 1, y - 1, pitch2, width, height) < centerPixel2) << 7;
        current_lbpCode2 |= (getPixel(d_img2, x, y - 1, pitch2, width, height) < centerPixel2) << 6;
        current_lbpCode2 |= (getPixel(d_img2, x + 1, y - 1, pitch2, width, height) < centerPixel2) << 5;
        current_lbpCode2 |= (getPixel(d_img2, x + 1, y, pitch2, width, height) < centerPixel2) << 4;
        current_lbpCode2 |= (getPixel(d_img2, x + 1, y + 1, pitch2, width, height) < centerPixel2) << 3;
        current_lbpCode2 |= (getPixel(d_img2, x, y + 1, pitch2, width, height) < centerPixel2) << 2;
        current_lbpCode2 |= (getPixel(d_img2, x - 1, y + 1, pitch2, width, height) < centerPixel2) << 1;
        current_lbpCode2 |= (getPixel(d_img2, x - 1, y, pitch2, width, height) < centerPixel2);

         __syncthreads();

        uint8_t diff = current_lbpCode1 ^ current_lbpCode2;
        double count = __popc(diff);

        d_similarities[y * pitch / sizeof(double) + x] = count / 8.0;
    }
}
